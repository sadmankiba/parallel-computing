#include "hip/hip_runtime.h"
#include "../src/linear.cuh"
#include "../src/relu.cuh"
#include "../src/softmax.cuh"
#include "../src/nllloss.cuh"
#include "../src/mnist.cuh"
#include "../src/base.cuh"
#include "../src/model.cuh"
#include "../src/utils.cuh"
#include <iostream>
#include <thrust/inner_product.h>
#include <ctime>
using namespace std;

struct MaxIndexFunctor {
    int numClasses;
    thrust::device_vector<float>::iterator logProbsStart;

    MaxIndexFunctor(int numClasses, thrust::device_vector<float>::iterator logProbsStart)
        : numClasses(numClasses), logProbsStart(logProbsStart) {}

    __device__ int operator()(int imageIndex) {
        auto start = logProbsStart + imageIndex * numClasses;
        auto end = start + numClasses;
        auto maxElement = thrust::max_element(thrust::seq, start, end);
        return thrust::distance(start, maxElement);
    }
};

float calcAccuracy(thrust::device_vector<float> &logProbs, thrust::device_vector<float> &labels, int numImages){
    int numClasses = 10;
    thrust::device_vector<int> predictions(numImages);
    thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(numImages), predictions.begin(), MaxIndexFunctor(numClasses, logProbs.begin()));

    int correctPredictions = thrust::inner_product(predictions.begin(), predictions.end(), labels.begin(), 0, thrust::plus<int>(), thrust::equal_to<int>());

    return (float)correctPredictions / numImages;
}

int main(int argc, char* argv[]){
     // timer setup
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // read data
    MNIST* mnist = new MNIST();
    int numImages = 60000;
    float* data = mnist->readData("train");
    thrust::device_vector<float> dataVector(data, data + numImages * 784);

    float* labels = mnist->readLabels("train");
    thrust::device_vector<float> labelsVector(numImages*10);
    oneHotEncodeLabels(labels, labelsVector, numImages, 10);

    // create model
    float lr = 1e-4;
    float beta = 0.99f;

    int numLayers = 8;
    std::vector<Base*> layers = std::vector<Base*>(numLayers);
    layers[0] = new Linear(784, 1024, lr, beta);
    layers[1] = new ReLU(1024);
    layers[2] = new Linear(1024, 512, lr, beta);
    layers[3] = new ReLU(512);
    layers[4] = new Linear(512, 256, lr, beta);
    layers[5] = new ReLU(256);
    layers[6] = new Linear(256, 10, lr, beta);
    layers[7] = new Softmax(10);

    Model model = Model(numLayers, layers);
    NLLLoss nllll = NLLLoss(10);

    // train model
    int numEpochs = 20;
    int batchSize = 64;
    int numBatches = (numImages + batchSize - 1) / batchSize;

    hipEventRecord(start);
    for (int i=0; i<numEpochs; i++){
        float cumBatchLoss = 0.0f;
        for (int j=0; j<numBatches; j++){
            int batchNumImages = min(batchSize, numImages - j*batchSize);
            thrust::device_vector<float> batchData(batchNumImages*784);
            thrust::device_vector<float>::iterator firstData = dataVector.begin() + j*batchSize*784;
            thrust::copy(firstData, firstData + batchNumImages*784, batchData.begin());

            thrust::device_vector<float> batchLabels(batchNumImages*10);
            thrust::device_vector<float>::iterator firstLabels = labelsVector.begin() + j*batchSize*10;
            thrust::copy(firstLabels, firstLabels + batchNumImages*10, batchLabels.begin());

            model.forward(batchData, batchNumImages);
            nllll.forward(model.output, batchLabels, batchNumImages);
            cumBatchLoss += nllll.loss;
            nllll.backward(batchNumImages);
            model.backward(nllll.gradient, batchNumImages);
        }
        cout << "epoch " << i << " avg batch loss: " << cumBatchLoss / numBatches << endl;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    cout << "training time for " << numEpochs << " epochs: " << ms << " ms" << endl;

    // test model
    int numTestImages = 10000;
    float* testData = mnist->readData("test");
    thrust::device_vector<float> testDataVector(testData, testData + numTestImages * 784);

    float* testLabels = mnist->readLabels("test");
    thrust::device_vector<float> testLabelsVector(testLabels, testLabels + numTestImages);

    hipEventRecord(start);
    model.forward(testDataVector, numTestImages);
    float acc = calcAccuracy(model.output, testLabelsVector, numTestImages);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    cout << "test accuracy: " << acc << endl;
    cout << "test time: " << ms << " ms" << endl;

    return 0;
}