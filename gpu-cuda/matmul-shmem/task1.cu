#include <hip/hip_runtime.h>
#include "matmul.cuh"
#include <iostream>
#include <cstdlib>
#include <ctime>

int main(int argc, char* argv[]){
    srand(time(0));
    
    unsigned int n = std::atoi(argv[1]);
    int N = n * n;
    unsigned int nBlock = std::atoi(argv[2]);

    // init int matrices
    int *Aint;
    int *Bint;
    int *Cint;
    hipMallocManaged((void **)&Aint, N * sizeof(int));
    hipMallocManaged((void **)&Bint, N * sizeof(int));
    hipMallocManaged((void **)&Cint, N * sizeof(int));
    for (int i=0; i< N; i++) {
        Aint[i] = (rand() % 3) - 0;
        Bint[i] = (rand() % 3) - 0;
    }

    // timer setup
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // execute matmul 1
    hipEventRecord(start);
    matmul_1(Aint, Bint, Cint, n, nBlock);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << Cint[0] << std::endl;
    std::cout << Cint[N-1] << std::endl;
    std::cout << ms << std::endl;
    hipFree(Cint);

    // init float matrices
    float *Afloat;
    float *Bfloat;
    float *Cfloat;
    hipMallocManaged((void **)&Afloat, N* sizeof(float));
    hipMallocManaged((void **)&Bfloat, N* sizeof(float));
    hipMallocManaged((void **)&Cfloat, N* sizeof(float));
    for (int i=0; i<N; i++) {
        Afloat[i] = static_cast<float>(Aint[i]);
        Bfloat[i] = static_cast<float>(Bint[i]);
    }

    // execute matmul 2
    hipEventRecord(start);
    matmul_2(Afloat, Bfloat, Cfloat, n, nBlock);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << Cfloat[0] << std::endl;
    std::cout << Cfloat[N-1] << std::endl;
    std::cout << ms << std::endl;
    hipFree(Cfloat);
    hipFree(Aint);
    hipFree(Bint);


    // init double matrices
    double *Adouble;
    double *Bdouble;
    double *Cdouble;
    hipMallocManaged((void **)&Adouble, N* sizeof(double));
    hipMallocManaged((void **)&Bdouble, N* sizeof(double));
    hipMallocManaged((void **)&Cdouble, N* sizeof(double));
    for (int i=0; i<N; i++) {
        Adouble[i] = static_cast<double>(Afloat[i]);
        Bdouble[i] = static_cast<double>(Bfloat[i]);
    }

    // execute matmul 3
    hipEventRecord(start);
    matmul_3(Adouble, Bdouble, Cdouble, n, nBlock);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    std::cout << Cdouble[0] << std::endl;
    std::cout << Cdouble[N-1] << std::endl;
    std::cout << ms << std::endl;

    // cleanup
    hipFree(Cdouble);
    hipFree(Afloat);
    hipFree(Bfloat);
    hipFree(Adouble);
    hipFree(Bdouble);

    return 0;
}
