#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>
#include "matmul.cuh"


float randomFloat(float min, float max){
    return min + (max-min)*(((float)rand())/(float)RAND_MAX);
}

int main(int argc, char* argv[]){
    srand(time(0));
    int n = std::atoi(argv[1]);
    int N = n*n;
    int tpb = std::atoi(argv[2]);

    // init host matrices
    float *A = new float[N];
    float *B = new float[N];
    float *C = new float[N];
    for (int i=0; i<N; i++) {
        A[i] = randomFloat(-1.0,1.0);
        B[i] = randomFloat(-1.0,1.0);
    }

    // init device matrices
    float *devA;
    float *devB;
    float *devC;
    hipMalloc((void**)&devA, sizeof(float)*(N));
    hipMalloc((void**)&devB, sizeof(float)*(N));
    hipMalloc((void**)&devC, sizeof(float)*(N));
    hipMemcpy(devA, A, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, sizeof(float)*N, hipMemcpyHostToDevice);

    // timer setup
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // execute
    hipEventRecord(start);
    matmul(devA, devB, devC, n, tpb);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // copy output back to host
    hipMemcpy(C, devC, sizeof(float)*N, hipMemcpyDeviceToHost);

    std::cout << C[N-1] << std::endl;
    std::cout << ms << std::endl;

    // cleanup
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}