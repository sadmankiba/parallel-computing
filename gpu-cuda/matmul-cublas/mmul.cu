#include "mmul.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>

// Uses a single cuBLAS call to perform the operation C := A B + C
// handle is a handle to an open cuBLAS instance
// A, B, and C are matrices with n rows and n columns stored in column-major
// NOTE: The cuBLAS call should be followed by a call to hipDeviceSynchronize() for timing purposes
void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n){
    hipblasStatus_t stat;
    const float alpha = 1.0f;
    const float beta = 0.0f;
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, C, n);

    hipDeviceSynchronize();

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d\n", stat);
    }
}