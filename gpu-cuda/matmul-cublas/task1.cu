#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>
#include "mmul.h"
#include <hipblas.h>

float randomFloat(float min, float max){
    return min + (max-min)*(((float)rand())/(float)RAND_MAX);
}

int main(int argc, char* argv[]){
    srand(time(0));
    int n = std::atoi(argv[1]);
    int N = n*n;
    int n_tests = std::atoi(argv[2]);

    // init matrices
    float *A;
    float *B;
    float *C;
    hipMallocManaged((void **)&A, N*sizeof(float));
    hipMallocManaged((void **)&B, N*sizeof(float));
    hipMallocManaged((void **)&C, N*sizeof(float));
    for (int i=0; i<N; i++) {
        A[i] = randomFloat(-1,1);
        B[i] = randomFloat(-1,1);
    }

    // init cuBLAS handle
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    // timer setup
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // execute
    float totalTime = 0;
    for (int i=0; i<n_tests; i++){
        hipEventRecord(start);
        mmul(handle, A, B, C, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        totalTime += ms;
    }

    float avg = totalTime/n_tests;
    std::cout << avg << std::endl;

    // cleanup
    hipFree(C);
    hipFree(A);
    hipFree(B);
    hipblasDestroy(handle);
    
    return 0;
}