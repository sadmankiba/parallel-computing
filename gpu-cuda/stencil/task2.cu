#include <iostream>
#include <stdlib.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stencil.cuh"

int main(int argc, char **argv)
{
    if (argc != 4)
    {
        std::cout << "Usage: ./task2 <n> <R> <threads_per_block>\n";
        exit(1);
    }
    int n = atoi(argv[1]);
    int R = atoi(argv[2]);
    int threads_per_block = atoi(argv[3]);
    
    float *image, *mask, *output;
    float *d_image, *d_mask, *d_output;

    // Allocate memory for each array on host
    image = (float *) malloc(n * sizeof(float));
    mask = (float *) malloc((2 * R + 1) * sizeof(float));
    output = (float *) malloc(n * sizeof(float));

    // Allocate memory for each array on GPU
    hipMalloc((void **)&d_image, n * sizeof(float));
    hipMalloc((void **)&d_mask, (2 * R + 1) * sizeof(float));
    hipMalloc((void **)&d_output, n * sizeof(float));

    srand(time(NULL));

    // Set image and mask array with random values in range [-1 ,1]
    for (int i = 0; i < n; i++){
        image[i] = (float) rand() / RAND_MAX * 2.0 - 1.0;
    }

    for (int i = 0; i < (2 * R + 1); i++) {
        mask[i] = (float) rand() / RAND_MAX * 2.0 - 1.0;
    }

    // Copy host arrays to device
    hipMemcpy(d_image, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, (2 * R + 1) * sizeof(float), hipMemcpyHostToDevice);
    
    // timer setup
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Run kernel
    hipEventRecord(start);
    stencil(d_image, d_mask, d_output, n, R, threads_per_block);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << output[n-1] << std::endl;
    std::cout << ms << std::endl;

    // Cleanup
    free(image);
    free(mask);
    free(output);

    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);

    return 0;
}