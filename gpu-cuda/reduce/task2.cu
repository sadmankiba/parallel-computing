#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <cmath>
#include "reduce.cuh"

int main(int argc, char **argv) {
    const unsigned int size = atoi(argv[1]);
    const unsigned int n_threads = atoi(argv[2]);  // threads per block
    float *input;
    float output;
    float *d_input, *d_output;

    srand(time(NULL));
    
    input = (float *) malloc(size * sizeof(float));

    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, ceil(size * 1.0 / n_threads) * sizeof(float));

    for (unsigned int i = 0; i < size; i++) {
        input[i] = rand() * 2.0 / RAND_MAX - 1.0;
    }

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);
    
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    reduce(&d_input, &d_output, size, n_threads);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(&output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << output << std::endl;
    std::cout << ms << std::endl;
}