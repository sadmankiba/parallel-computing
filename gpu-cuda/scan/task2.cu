#include <iostream>
#include <ctime>
#include <cmath>
#include "scan.cuh"

int main(int argc, char **argv) {
    const unsigned int size = atoi(argv[1]);
    const unsigned int n_threads = atoi(argv[2]);  // threads per block
    float *in, *out;

    srand(time(NULL));
    // srand(0);

    hipMallocManaged(&in, size * sizeof(float));
    hipMallocManaged(&out, size * sizeof(float));

    for (unsigned int i = 0; i < size; i++) {
        in[i] = rand() * 2.0 / RAND_MAX - 1.0;
        // in[i] = rand() % 3;
    }

    // std::cout << "in\n";
    // for (unsigned int i = 0; i < size; i++) {
    //     std::cout << in[i] << " ";
    // }
    // std::cout << std::endl;
    
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    scan(in, out, size, n_threads);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    // std::cout << "out\n";
    // for (unsigned int i = 0; i < size; i++) {
    //     std::cout << out[i] << " ";
    // }
    // std::cout << std::endl;

    std::cout << out[size - 1] << std::endl;
    std::cout << ms << std::endl;
}